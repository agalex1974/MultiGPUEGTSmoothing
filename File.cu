#include "hip/hip_runtime.h"
#include ""
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <vector>
__host__
float EucledianDistanceHost(int i, int j, float* pnts_x, float* pnts_y, float* pnts_z)
{
	float xi = pnts_x[i];
	float yi = pnts_y[i];
	float zi = pnts_z[i];
	float xj = pnts_x[j];
	float yj = pnts_y[j];
	float zj = pnts_z[j];
	return (xi - xj) * (xi - xj) + (yi - yj) * (yi - yj) + (zi - zj) * (zi - zj);
}

void cudaWarmup(float* points_x, float* points_y, float* points_z, int pointsCount, int k, int* kNN, int num_cards)
{
    for (int dev_id = 0; dev_id < num_cards; dev_id++) {
        hipSetDevice(dev_id);
        std::vector<float> distances(pointsCount);
        for (int i = 0; i < pointsCount; i++) {
            distances[i] = EucledianDistanceHost(k, i, points_x, points_y, points_z);
        }

        std::vector<int> distancesIdx(pointsCount);
        for (int i = 0; i < pointsCount; i++) {
            distancesIdx[i] = i;
        }

        float *ddistances;
        hipMalloc((void **) &ddistances, pointsCount * sizeof(float));
        hipMemcpy(ddistances, distances.data(), pointsCount * sizeof(float), hipMemcpyHostToDevice);

        int *ddistancesIdx;
        hipMalloc((void **) &ddistancesIdx, pointsCount * sizeof(int));
        hipMemcpy(ddistancesIdx, distancesIdx.data(), pointsCount * sizeof(int), hipMemcpyHostToDevice);

        thrust::sort_by_key(thrust::device, ddistances, ddistances + pointsCount, ddistancesIdx, thrust::less<float>());
        hipMemcpy(distancesIdx.data(), ddistancesIdx, pointsCount * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 1; i < 41; i++) {
            kNN[(i - 1) + k * 40] = distancesIdx[i];
        }

        hipFree(ddistances);
        hipFree(ddistancesIdx);
    }
}